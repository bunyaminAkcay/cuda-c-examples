
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

typedef float elem_t;

const int blockSize = 16;

void initRandomMatrix(elem_t *matrix, int n)
{
    for (size_t j = 0; j < n; j++)
    {
        for (size_t i = 0; i < n; i++)
        {
            matrix[n * j + i] = rand() / (elem_t)RAND_MAX;
        }
    }
}

void initZeroMatrix(elem_t *matrix, int n)
{
    for (size_t j = 0; j < n; j++)
    {
        for (size_t i = 0; i < n; i++)
        {
            matrix[n * j + i] = 0;
        }
    }
}

void printResult(elem_t *matrixA, elem_t *matrixB, elem_t *matrixC, int len)
{

    elem_t *matrixReferences[3] = {matrixA, matrixB, matrixC};
    char matrixNames[3] = {'A', 'B', 'C'};
    for (size_t k = 0; k < 3; k++)
    {
        printf("%c [\n\n\t", matrixNames[k]);
        for (size_t j = 0; j < len; j++)
        {
            for (size_t i = 0; i < len; i++)
            {
                printf("%f ", matrixReferences[k][len * j + i]);
            }
            printf("\n\t");
        }
        printf("\n]\n");
    }
}

__global__ void matMulWithTilingKernel(elem_t *matrixA_d, elem_t *matrixB_d, elem_t *matrixC_d, int n){
    __shared__ elem_t Atile[blockSize][blockSize];
    __shared__ elem_t Btile[blockSize][blockSize];
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    elem_t value = 0;
    int numTiles = (n + blockSize - 1) / blockSize;
    for (size_t j = 0; j < numTiles; j++){
        if (row < n && (j * blockSize + tx) < n){
            Atile[ty][tx] = matrixA_d[row * n + j * blockSize + tx];
        }
        else{
            Atile[ty][tx] = 0;
        }

        if ((j * blockSize + ty) < n && col < n){
            Btile[ty][tx] = matrixB_d[(j * blockSize + ty) * n + col];
        }
        else{
            Btile[ty][tx] = 0;
        }
        __syncthreads();
        for (size_t k = 0; k < blockSize; k++)
        {
            value += Atile[ty][k] * Btile[k][tx];
        }
        __syncthreads();
    }
    if ((row < n) && (col < n)){
        matrixC_d[row * n + col] = value;
    }
}

void matMulWithTiling(elem_t *matrixA_h, elem_t *matrixB_h, elem_t *matrixC_h, int n)
{

    elem_t *matrixA_d, *matrixB_d, *matrixC_d;
    size_t size = n * n * sizeof(elem_t);

    hipMalloc((void **) &matrixA_d, size);
    hipMalloc((void **) &matrixB_d, size);
    hipMalloc((void **) &matrixC_d, size);

    hipMemcpy(matrixA_d, matrixA_h, n * n * sizeof(elem_t), hipMemcpyHostToDevice);
    hipMemcpy(matrixB_d, matrixB_h, n * n * sizeof(elem_t), hipMemcpyHostToDevice);

    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid((n + blockSize - 1) / blockSize, (n + blockSize - 1) / blockSize, 1);

    matMulWithTilingKernel<<<dimGrid, dimBlock>>>(matrixA_d, matrixB_d, matrixC_d, n);

    hipMemcpy(matrixC_h, matrixC_d, n * n * sizeof(elem_t), hipMemcpyDeviceToHost);

    hipFree(matrixA_d);
    hipFree(matrixB_d);
    hipFree(matrixC_d);
}

int main()
{
    int n = 512;

    elem_t *matrixA = (elem_t *)malloc(n * n * sizeof(elem_t));
    elem_t *matrixB = (elem_t *)malloc(n * n * sizeof(elem_t));
    elem_t *matrixC = (elem_t *)malloc(n * n * sizeof(elem_t));

    initRandomMatrix(matrixA, n);
    initRandomMatrix(matrixB, n);
    initZeroMatrix(matrixC, n);

    printf("MatMul with tiling:\n");
    matMulWithTiling(matrixA, matrixB, matrixC, n);

    //printResult(matrixA, matrixB, matrixC, n);

    free(matrixA);
    free(matrixB);
    free(matrixC);

    return 0;
}