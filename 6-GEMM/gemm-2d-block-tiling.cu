
#include <hip/hip_runtime.h>
#include <cstdio>

typedef float element_t;

template <typename T>
__host__ __device__ T blockCeil(T const n, T const blockSize){
    return (n + blockSize -1)/blockSize;
}

/*
    C = alpha * AB + beta * C

    dimension of A is m * k
    dimension of B is k * n
    dimension of C is m * n
*/


/*
    BLOCK_TILE_X, BLOCK_TILE_K, BLOCK_TILE_Y is given
    as templete parameter because these are compile-time
    constant. If we use blockDim.x, blockDim.y instead
    of that, we cannot unroll loops due to blockDim.x
    and blockDim.y is run-time constant.
*/
template <typename T, size_t BLOCK_TILE_Y, size_t BLOCK_TILE_K, size_t BLOCK_TILE_X>
__global__ void gemm2dBlockTilingKernel(size_t const m,
                                        size_t const n,
                                        size_t const k,
                                        T const alpha,
                                        T const beta,
                                        T const* A,
                                        T const* B,
                                        T* C)
{
    //cCol is the x position of thread
    size_t const cColIdx = blockIdx.x * blockDim.x + threadIdx.x;
    //cRow is the y position of thread
    size_t const cRowIdx = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ T aTile[BLOCK_TILE_Y][BLOCK_TILE_K];
    __shared__ T bTile[BLOCK_TILE_K][BLOCK_TILE_X];

    size_t const numberOfTiles = blockCeil(k, BLOCK_TILE_K);
    T value = static_cast<T>(0);
    for (size_t tileId = 0; tileId < numberOfTiles; tileId++)
    {
        //load data from dram to shared memory
        if (cRowIdx < m && (tileId * BLOCK_TILE_K + threadIdx.x) < k){
            aTile[threadIdx.y][threadIdx.x] = A[cRowIdx * k + tileId * BLOCK_TILE_K + threadIdx.x];
        }
        else{
            aTile[threadIdx.y][threadIdx.x] = 0;
        }

        if ((tileId * BLOCK_TILE_K + threadIdx.y) < k && cColIdx < n){
            bTile[threadIdx.y][threadIdx.x] = B[(tileId * BLOCK_TILE_K + threadIdx.y) * n + cColIdx];
        }
        else{
            bTile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        
        //add to value
        #pragma unroll
        for (size_t ki = 0; ki < BLOCK_TILE_K; ki++)
        {
            value += aTile[threadIdx.y][ki] * bTile[ki][threadIdx.x];
        }
        __syncthreads();
    }

    if (cRowIdx < m && cColIdx < n){
        C[cRowIdx * n + cColIdx] = alpha * value + beta * C[cRowIdx * n + cColIdx];
    }

}

template<typename T>
void gemm2dBlockTiling( size_t m,
                size_t n,
                size_t k,
                T const alpha,
                T const beta,
                T const* h_A,
                T const* h_B,
                T const* h_C,
                T *h_D)
{
    T *d_A, *d_B, *d_C;
    size_t const blockWidth = 32;

    hipMalloc((void**) &d_A, m * k * sizeof(T));
    hipMalloc((void**) &d_B, k * n * sizeof(T));
    hipMalloc((void**) &d_C, m * n * sizeof(T));

    hipMemcpy(d_A, h_A, m * k * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, m * n * sizeof(T), hipMemcpyHostToDevice);

    dim3 const dimBlock(blockWidth, blockWidth, 1U);
    dim3 const dimGrid(blockCeil(static_cast<uint>(n), dimBlock.x), blockCeil(static_cast<uint>(m), dimBlock.y), 1U);

    gemm2dBlockTilingKernel<T, blockWidth, blockWidth, blockWidth><<<dimGrid, dimBlock>>>(m, n, k, alpha, beta, d_A, d_B, d_C);

    hipMemcpy(h_D, d_C, m * n * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

template<typename T>
void initRandomMatrix(T *matrix, size_t const rowCount, size_t const colCount)
{
    for (size_t row = 0; row < rowCount; row++)
    {
        for (size_t col = 0; col < colCount; col++)
        {
            matrix[row * colCount + col] = rand() / static_cast<T>(RAND_MAX);
        }
    }
}

template<typename T>
void initZeroMatrix(T *matrix, size_t const rowCount, size_t const colCount)
{
    for (size_t row = 0; row < rowCount; row++)
    {
        for (size_t col = 0; col < colCount; col++)
        {
            matrix[row * colCount + col] = 0;
        }
    }
}

template<typename T>
void printResult(size_t const m, size_t const n, size_t const k, T const* A, T const* B, T const* C, T const* D)
{
    size_t const matrixCount = 4;
    T const* matrixReferences[matrixCount] = {A, B, C, D};
    
    char matrixNames[matrixCount] = {'A', 'B', 'C', 'D'};
    size_t matrixDimensions[matrixCount][2] = { {m, k}, {k, n}, {m, n}, {m, n}};

    for (size_t k = 0; k < matrixCount; k++)
    {
        printf("%c [\n\n\t", matrixNames[k]);
        for (size_t j = 0; j < matrixDimensions[k][0]; j++)
        {
            for (size_t i = 0; i < matrixDimensions[k][1]; i++)
            {
                printf("%f ", matrixReferences[k][matrixDimensions[k][1] * j + i]);
            }
            printf("\n\t");
        }
        printf("\n]\n");
    }
}

int main(){

    bool const printResults = false;

    size_t m = 1024, k = 1024, n = 1024;
    element_t alpha = 1, beta = 0;

    element_t *h_A = (element_t *)malloc(m * k * sizeof(element_t));
    element_t *h_B = (element_t *)malloc(k * n * sizeof(element_t));
    element_t *h_C = (element_t *)malloc(m * n * sizeof(element_t));
    element_t *h_D = (element_t *)malloc(m * n * sizeof(element_t));

    initRandomMatrix<element_t>(h_A, m, k);
    initRandomMatrix<element_t>(h_B, k, n);
    initZeroMatrix<element_t>(h_C, m, n);
    initZeroMatrix<element_t>(h_D, m, n);
    
    gemm2dBlockTiling<element_t>(m, n, k, alpha, beta, h_A, h_B, h_C, h_D);

    if (printResults)
    {
        printResult<element_t>(m, n, k, h_A, h_B, h_C, h_D);
    }
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    return 0;
}