#include "MatrixParser.h"
#include "MatrixHelper.h"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

typedef float element_t;

void cuSparseSpMMwithCOO(int* h_A_rowIds, int* h_A_colIds, element_t* h_A_values, element_t* h_B, element_t* h_C, element_t* h_D, size_t m, size_t k, size_t n, size_t nnz, element_t alpha, element_t beta){
    
    element_t *d_B, *d_C;
    element_t *d_A_colIds, *d_A_rowIds, *d_A_values; 

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipMalloc((void**) &d_B, k * n * sizeof(element_t));
    hipMalloc((void**) &d_C, m * n * sizeof(element_t));
    hipMalloc((void**) &d_A_colIds, nnz * sizeof(int));
    hipMalloc((void**) &d_A_rowIds, nnz * sizeof(int));
    hipMalloc((void**) &d_A_values, nnz * sizeof(int));

    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA memory allocation failed\n");
        return;
    }

    hipMemcpy(d_B, h_B, k * n * sizeof(element_t), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, m * n * sizeof(element_t), hipMemcpyHostToDevice);
    hipMemcpy(d_A_colIds, h_A_colIds, nnz * sizeof(element_t), hipMemcpyHostToDevice);
    hipMemcpy(d_A_rowIds, h_A_rowIds, nnz * sizeof(element_t), hipMemcpyHostToDevice);
    hipMemcpy(d_A_values, h_A_values, nnz * sizeof(element_t), hipMemcpyHostToDevice);

    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "CUDA memory copy failed\n");
        return;
    }

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    hipsparseCreateCoo(&matA, m, k, nnz, d_A_rowIds, d_A_colIds, d_A_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    hipsparseCreateDnMat(&matB, k, n, n, d_B, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&matC, m, n, n, d_C, HIP_R_32F, HIPSPARSE_ORDER_ROW);

    size_t bufferSize;
    void *dBuffer;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Perform SpMM
    hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);


    hipMemcpy(h_D, d_C, m * n * sizeof(element_t), hipMemcpyDeviceToHost);

    // Free workspace
    

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);

    hipFree(dBuffer);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_colIds);
    hipFree(d_A_rowIds);
    hipFree(d_A_values);

}

int main()
{
    bool symmetrical = false;
    bool const printResults = true;
    
    
    CooMatrixParser<element_t> cooMatrixParser("matrices/b1_ss.mtx", symmetrical, Order::rowMajor);
    
    
    cooMatrixParser.saveSparseMatrixAsPPM3Image("matrixImages/b1_ss");
    
    size_t m = cooMatrixParser.getRowCount();
    size_t k = cooMatrixParser.getColCount();
    size_t n = m;
    size_t nnz = cooMatrixParser.getNNZ();

    element_t alpha = static_cast<element_t>(1);
    element_t beta = static_cast<element_t>(0);

    int* h_A_rowIds = cooMatrixParser.rowIds;
    int* h_A_colIds = cooMatrixParser.colIds;
    element_t* h_A_values = cooMatrixParser.values; 

    element_t *h_B = (element_t *)malloc(k * n * sizeof(element_t));
    element_t *h_C = (element_t *)malloc(m * n * sizeof(element_t));
    element_t *h_D = (element_t *)malloc(m * n * sizeof(element_t));
    
    MatrixHelper<element_t>::initRandomDenseMatrix(h_B, k, n);
    MatrixHelper<element_t>::initZeroMatrix(h_C, m, n);
    MatrixHelper<element_t>::initZeroMatrix(h_D, m, n);
    
    cuSparseSpMMwithCOO(h_A_rowIds, h_A_colIds, h_A_values, h_B, h_C, h_D, m, k, n, nnz, alpha, beta);

    if(printResults && cooMatrixParser.sparseMatrixToClassicMatrix()){
        MatrixHelper<element_t>::printResult(m, n, k, cooMatrixParser.getClassicMatrix(), h_B, h_C, h_D);
    } 
    
    free(h_B);
    free(h_C);
    free(h_D);
    
    return 0;
}